#include "hip/hip_runtime.h"
#include "assert.h"
#include "raycast.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include <math.h>
#include "math.cuh"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

typedef struct {
    vec3 origin;
    vec3 dir;
} Ray;

typedef struct {
    vec3 point;
    vec3 normal;
    Material material;
    float t;
    bool front_face;
} HitRecord;

__global__ void setup_rng(hiprandState* state, int width, int height, int seed) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;

    int idx = y * width + x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

__device__ bool range_constains(float v, float min, float max) {
    return min <= v && v <= max;
}

__device__ bool range_surrounds(float v, float min, float max) {
    return min < v && v < max;
}

__device__ vec3 ray_at(const Ray* ray, float t) {
    return ray->origin + t * ray->dir;
}

__device__ float color_linear_to_gamma(float comp) {
    if (comp > 0.0f) return sqrtf(comp);
    return 0.0f;
}

__device__ bool scatter_lambertian(const Ray* ray, const HitRecord* hit_record, hiprandState* local_state, vec3* attenuation, Ray* scattered) {
    vec3 scatter_dir = hit_record->normal + random_unit_vector(local_state);
    if (near_zero(scatter_dir)) {
        scatter_dir = hit_record->normal;
    }

    scattered->dir = scatter_dir;
    attenuation->x = attenuation->x * hit_record->material.albedo.x;
    attenuation->y = attenuation->y * hit_record->material.albedo.y;
    attenuation->z = attenuation->z * hit_record->material.albedo.z;
    return true;
}

__device__ bool scatter_metal(const Ray* ray, const HitRecord* hit_record, hiprandState* local_state, vec3* attenuation, Ray* scattered) {
    vec3 reflected = reflect(ray->dir, hit_record->normal);
    reflected = normalize(reflected) + (hit_record->material.fuzz * random_unit_vector(local_state));
    scattered->dir = reflected;
    attenuation->x = attenuation->x * hit_record->material.albedo.x;
    attenuation->y = attenuation->y * hit_record->material.albedo.y;
    attenuation->z = attenuation->z * hit_record->material.albedo.z;
    return dot(scattered->dir, hit_record->normal) > 0.0f;
}

__device__ bool sphere_hit(const Sphere *sphere, const Ray *ray, float ray_tmin,
                           float ray_tmax, HitRecord *hit_record) {
    const vec3 oc = sphere->center - ray->origin;
    const float a = dot(ray->dir, ray->dir);
    const float h = dot(ray->dir, oc);
    const float c = dot(oc, oc) - sphere->radius * sphere->radius;
    const float disc = h * h - a * c;
    if (disc < 0.0f) {
        return false;
    }

    const float sqrtd = sqrtf(disc);
    float root = (h - sqrtd) / a;
    if (!range_surrounds(root, ray_tmin, ray_tmax)) {
        root = (h + sqrtd) / a;
        if (!range_surrounds(root, ray_tmin, ray_tmax)) {
            return false;
        }
    }

    const vec3 point = ray_at(ray, root);
    hit_record->t = root;
    hit_record->point = point;

    const vec3 outward_normal = (point - sphere->center) / sphere->radius;
    hit_record->front_face = dot(ray->dir, outward_normal) < 0.0;
    hit_record->normal = hit_record->front_face ? outward_normal : outward_normal * -1.0f;

    return true;
}

__device__ bool spheres_hit(const Ray* ray, const Sphere *spheres, unsigned int spheres_count, float ray_tmin, float ray_tmax, HitRecord* hit_record) {
    bool hit = false;
    float closest_so_far = ray_tmax;
    for (size_t i = 0u; i < spheres_count; ++i) {
        HitRecord temp_hit = {};
        const Sphere* sphere = &spheres[i];
        bool _hit = sphere_hit(sphere, ray, ray_tmin, closest_so_far, &temp_hit);
        if (_hit) {
            hit = true;
            closest_so_far = temp_hit.t;
            hit_record->t = temp_hit.t;
            hit_record->material = sphere->material;
            hit_record->normal = temp_hit.normal;
            hit_record->point = temp_hit.point;
            hit_record->front_face = temp_hit.front_face;
        }
    }

    return hit;
}

__device__ vec3 sample_square(hiprandState *local_state) {
    float x = hiprand_uniform(local_state) - 0.5f;
    float y = hiprand_uniform(local_state) - 0.5f;
    float z = 0.0f;
    return {x, y, z};
}

__device__ vec3 ray_color(const Ray& ray, int max_depth, const Sphere* spheres, unsigned int spheres_count, hiprandState* local_state) {
    Ray current_ray = ray;
    vec3 attenuation = {1.0f, 1.0f, 1.0f};
    vec3 color = {0.0f, 0.0f, 0.0f};

    for (int depth = 0; depth < max_depth; ++depth) {
        HitRecord hit_record;
        if (spheres_hit(&current_ray, spheres, spheres_count, 0.001f, INFINITY, &hit_record)) {
            bool scattered = false;
            Ray temp_ray = {current_ray.origin, current_ray.dir};
            switch (hit_record.material.kind) {
            case MAT_LAMBERTIAN:
                scattered = scatter_lambertian(&current_ray, &hit_record, local_state, &attenuation, &temp_ray);
                break;
            case MAT_METAL:
                scattered = scatter_metal(&current_ray, &hit_record, local_state, &attenuation, &temp_ray);
                break;
            case MAT_EMISSIVE:
                return color + attenuation * hit_record.material.emit;
            }
            if (scattered) {
                current_ray.origin = hit_record.point + 1e-4f * hit_record.normal;
                current_ray.dir = temp_ray.dir;
            } else {
                color = {0.0f, 0.0f, 0.0f};
                break;
            }
        } else {
            const vec3 unit_dir = normalize(current_ray.dir);
            float t = 0.5f * (unit_dir.y + 1.0f);
            color = color + ((1.0f - t) * vec3{1.0f, 1.0f, 1.0f} + t * vec3{0.5f, 0.7f, 1.0f}) * attenuation;
            break;
        }
    }
    return color;
}

__global__ void render_kernel(unsigned char* img, const CameraData* cam, const Sphere* spheres, unsigned int spheres_count, hiprandState* rng_state) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= cam->image_width || y >= cam->image_height) return;

    const float viewport_height = 2.0f;
    const float viewport_width  = viewport_height * (float)cam->image_width / (float)cam->image_height;

    const vec3 pixel_delta_u = {viewport_width / (float)cam->image_width, 0.0f, 0.0f};
    const vec3 pixel_delta_v = {0.0f, -viewport_height / (float)cam->image_height, 0.0f};

    const vec3 viewport_upper_left = {-viewport_width / 2.0f, viewport_height / 2.0f, -cam->focal_length};
    const vec3 pixel00_loc = viewport_upper_left + (pixel_delta_u + pixel_delta_v) * 0.5f;

    hiprandState* local_state = &rng_state[y * cam->image_width + x];
    vec3 color = {0.0f, 0.0f, 0.0f};

    for (size_t sample = 0u; sample < cam->samples_per_pixel; ++sample) {
        vec3 offset = sample_square(local_state);
        vec3 pixel_sample = pixel00_loc + (x + offset.x) * pixel_delta_u + (y + offset.y) * pixel_delta_v;

        float4 origin_cam = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
        float4 dir_cam    = make_float4(pixel_sample.x, pixel_sample.y, pixel_sample.z, 0);

        float4 origin_world4 = mmul(cam->camera_to_world, origin_cam);
        float4 dir_world4    = mmul(cam->camera_to_world, dir_cam);

        Ray ray = Ray {
            .origin = vec3{origin_world4.x, origin_world4.y, origin_world4.z},
            .dir    = normalize({dir_world4.x, dir_world4.y, dir_world4.z}),
        };

        color = color + ray_color(ray, cam->max_depth, spheres, spheres_count, local_state);
    }

    color = color / (float)cam->samples_per_pixel;

    // gamma correction
    float r = color_linear_to_gamma(color.x);
    float g = color_linear_to_gamma(color.y);
    float b = color_linear_to_gamma(color.z);

    int idx = 3 * (y * cam->image_width + x);
    img[idx+0] = (unsigned char)(255.0f * clamp(r, 0.0f, 0.999f));
    img[idx+1] = (unsigned char)(255.0f * clamp(g, 0.0f, 0.999f));
    img[idx+2] = (unsigned char)(255.0f * clamp(b, 0.0f, 0.999f));
}

Sphere* d_spheres;
unsigned char *d_img;
hiprandState *d_rng_state;


EXTERN_C VertexBuffer* vb_alloc(size_t count) {
    // Allocate the struct on the host heap
    VertexBuffer *vb = (VertexBuffer*)malloc(sizeof(VertexBuffer));
    assert(vb != NULL);

    // Allocate device memory for the buffers
    hipMalloc((void**)&vb->p_buf, count * sizeof(vec3));
    hipMalloc((void**)&vb->c_buf, count * sizeof(vec3));
    hipMalloc((void**)&vb->n_buf, count * sizeof(vec3));
    vb->count = count;

    hipDeviceSynchronize();

    return vb;
}

EXTERN_C void vb_free(VertexBuffer *vb) {
    assert(vb != NULL);

    hipFree(vb->p_buf);
    hipFree(vb->c_buf);
    hipFree(vb->n_buf);
    hipDeviceSynchronize();

    free(vb);
}

EXTERN_C void init_cuda(const CameraData *cam, size_t spheres_count, int seed) {
    size_t img_size = cam->image_height * cam->image_width * 3U * sizeof(unsigned char);
    hipMalloc((void**)&d_spheres, spheres_count * sizeof(Sphere));
    hipMalloc((void**)&d_img, img_size);

    hipMalloc(&d_rng_state, cam->image_height * cam->image_width * sizeof(hiprandState));

    dim3 block(16, 16);
    dim3 grid((cam->image_width + block.x - 1) / block.x,
                (cam->image_height + block.y - 1) / block.y);

    setup_rng<<<grid, block>>>(d_rng_state, cam->image_width, cam->image_height, seed);

    hipDeviceSynchronize();
}

EXTERN_C void update_spheres(const Sphere *spheres, size_t spheres_count) {
    hipMemcpy(d_spheres, spheres, spheres_count * sizeof(spheres[0]), hipMemcpyHostToDevice);
}

EXTERN_C void launch_raycast(unsigned char *img, const CameraData* cam, const Sphere* spheres, size_t spheres_count) {
    size_t img_size = cam->image_height * cam->image_width * 3U * sizeof(unsigned char);
    dim3 block(16, 16);
    dim3 grid((cam->image_width + block.x - 1) / block.x,
                (cam->image_height + block.y - 1) / block.y);

    render_kernel<<<grid, block>>>(d_img, cam, d_spheres, spheres_count, d_rng_state);

    hipMemcpy(img, d_img, img_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
}

EXTERN_C void cleanup_cuda(void) {
    hipFree(d_img);
    hipFree(d_rng_state);
    hipFree(d_spheres);
}
